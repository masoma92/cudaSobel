#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include <math.h>
#include "res\Timing.h"

#define IMG_INPUT "img\\input.bmp"
#define IMG_OUTPUT "img\\output.bmp"
#define IMG_OUTPUT2 "img\\output2.bmp"
#define IMG_HEADER 1078
#define IMG_WIDTH 4000
#define IMG_HEIGHT 4000

//minden block 2 dimenziós lesz, 32*32 így minden block 1024 szálat tud futtatni egy időben
//ez annyit jelent, hogy egy 4000*4000-es képnél 125 block vízszintesen, 125 függőlegesen

__device__ char* dev_origin;
__device__ char* dev_result;

__global__ void EdgeDetect(int width, int height) {

	//ez felel meg a szekvenciális kódban a két egybeágyazott for ciklusnak
	int row = blockIdx.y * blockDim.y + threadIdx.y; //i blockidx a hanyadik block az oszlopban, blockdim az a blokkon belüli sor
	int col = blockIdx.x * blockDim.x + threadIdx.x; //j


	if (row >= height || col >= width || row < 1 || col < 1) return; //olyan szál le se fusson ami nem a képen kívül van

	int Gx[3][3] = { {-1,0,1}, {-2,0,2}, {-1,0,1} };
	int Gy[3][3] = { {1,2,1}, {0,0,0}, {-1,-2,-1} };

	int sumX, sumY;
	sumX = sumY = 0;

	for (int i = -1; i <= 1; i++) {
		for (int j = -1; j <= 1; j++) {
			int curPixel = dev_origin[(row + j) * width + (col + i)];
			sumX += curPixel * Gx[i + 1][j + 1];
			sumY += curPixel * Gy[i + 1][j + 1];
		}
	}

	int sum = abs(sumY) + abs(sumX);
	if (sum > 255) sum = 255;
	if (sum < 0) sum = 0;

	dev_result[row * width + col] = sum;
}


int main()
{
	FILE* f_input_img, * f_output_img;
	unsigned char* img;
	unsigned char* out_img;

	// Allocate CUDA managed host memory
	hipHostMalloc((void**)&img, IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT, 0);
	hipHostMalloc((void**)&out_img, IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT, 0);

	// Load header
	fopen_s(&f_input_img, IMG_INPUT, "rb");
	fread(img, 1, IMG_HEADER, f_input_img);

	//ToDo: Allocate device memory

	hipMalloc((void**)&img, IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT);
	hipMemcpy((void*)&dev_origin, &img, IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT, hipMemcpyHostToDevice);

	hipMemcpy((void**)&dev_result, &img, IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT, hipMemcpyHostToDevice);

	EdgeDetect << <dim3(125 * 125), dim3(32 * 32) >> > (IMG_WIDTH, IMG_HEIGHT);

	hipMemcpy((void**)&img, &dev_result, IMG_HEADER + sizeof(unsigned char) * IMG_WIDTH * IMG_HEIGHT, hipMemcpyDeviceToHost);

		// Save file
	fopen_s(&f_output_img, IMG_OUTPUT, "wb");
	fwrite(img, 1, IMG_HEADER + IMG_WIDTH * IMG_HEIGHT, f_output_img);
	fclose(f_output_img);

	//ToDo: Free device memory

		// Free CUDA managed host memory
	hipHostFree(img);
}
